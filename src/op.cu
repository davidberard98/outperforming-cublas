#include "hip/hip_runtime.h"
#include "ATen/ATen.h"
#include "torch/extension.h"

#include <hip/hip_runtime.h>
#include <cuda/barrier>
#include <cudaTypedefs.h>
#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>

#include <cassert>

using bf16 = __hip_bfloat16;

#include "mm_v1.cuh"

TORCH_LIBRARY(gemm, m) { m.def("mm_v1(Tensor a, Tensor b) -> Tensor", &mm_v1); }
